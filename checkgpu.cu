// -*- c++ -*-
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void initDevice(int argc, char **argv)
{
  hipError_t ret;
  int devicenum = -1;
  int i;
  for(i=1; i<argc; i++){
    if(strncmp(argv[i], "-devicenum", 10)==0){
      devicenum = atoi(argv[++i]);
      printf("devicenum = %d\n", devicenum);
    }
  }

  //cudaSetDeviceFlags(cudaDeviceMapHost);

  if(devicenum==-1){
    int nMaxDevices=-1;
    int nDevice=-1;
    int nMajor=-1, nMinor=-1;
    hipDeviceProp_t deviceProp;
    ret = hipGetDeviceCount(&nMaxDevices);
    if(ret!=hipSuccess){
      printf("hipGetDeviceCount failed, exit\n");
      exit(-1);
    }
    printf("%d device(s) found\n", nMaxDevices);
    for(i=0; i<nMaxDevices; i++){
      hipGetDeviceProperties(&deviceProp, i);
      printf("GPU %d: %s, has %d processors\n", i, deviceProp.name, deviceProp.multiProcessorCount);
      if(deviceProp.major > nMajor){
	nDevice=i;
	nMajor = deviceProp.major;
	nMinor = deviceProp.minor;
      }else if(deviceProp.major == nMajor){
	if(deviceProp.minor > nMinor){
	  nDevice=i;
	  nMajor = deviceProp.major;
	  nMinor = deviceProp.minor;
	}
      }
    }
    hipGetDeviceProperties(&deviceProp, nDevice);
    printf("use %d: %s, has %d processors\n", nDevice, deviceProp.name, deviceProp.multiProcessorCount);
    hipSetDevice(nDevice);
  }else{
    int nMaxDevices=-1;
    int nDevice=devicenum;
    hipDeviceProp_t deviceProp;
    ret = hipGetDeviceCount(&nMaxDevices);
    if(ret!=hipSuccess){
      printf("hipGetDeviceCount failed, exit\n");
      exit(-1);
    }
    printf("%d device(s) found\n", nMaxDevices);
    for(i=0; i<nMaxDevices; i++){
      hipGetDeviceProperties(&deviceProp, i);
      printf("GPU %d: %s, has %d processors\n", i, deviceProp.name, deviceProp.multiProcessorCount);
    }
    hipGetDeviceProperties(&deviceProp, nDevice);
    printf("use %d: %s, has %d processors\n", nDevice, deviceProp.name, deviceProp.multiProcessorCount);
    hipSetDevice(nDevice);
  }
  hipDeviceReset();
  printf("initDevice: done\n"); fflush(stdout);
}

int main(int argc, char **argv)
{
  initDevice(argc, argv);
  return 0;
}

